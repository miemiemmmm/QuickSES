#include "hip/hip_runtime.h"
/*MIT License

Copyright (c) 2019 Xavier Martinez

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.
*/
#include <algorithm>
#include <cctype>
#include <fstream>
#include <functional>
#include <iostream>
#include <iterator>
#include <locale>
#include <math.h>
#include <map>
#include <memory>
#include <stdlib.h>
#include <stdio.h>
#include <sstream>
#include <string.h>
#include <vector>

#include <cpdb/cpdb.h>
#include "args.hxx"
#include "Kernels.cu"
#include "SmoothMesh.h"
#include "CudaSurf.h"
#include "ObjFormats.h"

#include "hip/hip_runtime.h"
#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/sequence.h>
#include <thrust/binary_search.h>
#include <thrust/sort.h>
#include <thrust/unique.h>
#include <thrust/count.h>

using namespace std;

int SLICE = 300;
float probeRadius = PROBERADIUS;
float gridResolutionNeighbor;
float gridResolutionSES = 0.5f;
int laplacianSmoothSteps = 1;
string outputFilePath = "output.obj";
string inputFilePath = "";
std::map<char, float> radiusDic;
void initRadiusDic() {
    float factor = 1.0f;
    radiusDic['O'] = 1.52f * factor;
    radiusDic['C'] = 1.70f * factor;
    radiusDic['N'] = 1.55f * factor;
    radiusDic['H'] = 1.20f * factor;
    radiusDic['S'] = 2.27f * factor;
    radiusDic['P'] = 1.80f * factor;
    radiusDic['X'] = 1.40f * factor;
}


unsigned int getMinMax(chain *C, float3 *minVal, float3 *maxVal, float *maxAtom) {
    atom *A = NULL;
    unsigned int N = 0;


    A = &C->residues[0].atoms[0];
    float3 vmin, vmax, coords;

    vmin.x = vmin.y = vmin.z = 100000.0f;
    vmax.x = vmax.y = vmax.z = -100000.0f;
    *maxAtom = 0.0f;
    while (A != NULL) {
        coords = A->coor;
        vmin.x = std::min(vmin.x, coords.x);
        vmin.y = std::min(vmin.y, coords.y);
        vmin.z = std::min(vmin.z, coords.z);

        vmax.x = std::max(vmax.x, coords.x);
        vmax.y = std::max(vmax.y, coords.y);
        vmax.z = std::max(vmax.z, coords.z);

        float atomRad;
        if (radiusDic.count(A->element[0]))
            atomRad = radiusDic[A->element[0]];
        else
            atomRad = radiusDic['X'];
        *maxAtom = std::max(*maxAtom, atomRad);
        N++;
        A = A->next;
    }
    *minVal = vmin;
    *maxVal = vmax;
    return N;
}


unsigned int getMinMax(pdb *P, float3 *minVal, float3 *maxVal, float *maxAtom) {
    atom *A = NULL;
    unsigned int N = 0;
    chain *C = NULL;
    *maxAtom = 0.0f;
    float3 vmin, vmax, coords;

    vmin.x = vmin.y = vmin.z = 100000.0f;
    vmax.x = vmax.y = vmax.z = -100000.0f;

    for (int chainId = 0; chainId < P->size; chainId++) {
        C = &P->chains[chainId];

        A = &C->residues[0].atoms[0];

        while (A != NULL) {
            coords = A->coor;
            vmin.x = std::min(vmin.x, coords.x);
            vmin.y = std::min(vmin.y, coords.y);
            vmin.z = std::min(vmin.z, coords.z);

            vmax.x = std::max(vmax.x, coords.x);
            vmax.y = std::max(vmax.y, coords.y);
            vmax.z = std::max(vmax.z, coords.z);

            float atomRad;
            if (radiusDic.count(A->element[0]))
                atomRad = radiusDic[A->element[0]];
            else
                atomRad = radiusDic['X'];
            *maxAtom = std::max(*maxAtom, atomRad);
            N++;
            A = A->next;
        }
    }
    *minVal = vmin;
    *maxVal = vmax;
    return N;
}


void getMinMax(float3 *positions, float *radii, unsigned int N, float3 *minVal, float3 *maxVal, float *maxAtom) {
    *maxAtom = 0.0f;
    float3 vmin, vmax, coords;

    vmin.x = vmin.y = vmin.z = 100000.0f;
    vmax.x = vmax.y = vmax.z = -100000.0f;

    for (unsigned int a = 0; a < N; a++) {
        coords = positions[a];
        vmin.x = std::min(vmin.x, coords.x);
        vmin.y = std::min(vmin.y, coords.y);
        vmin.z = std::min(vmin.z, coords.z);

        vmax.x = std::max(vmax.x, coords.x);
        vmax.y = std::max(vmax.y, coords.y);
        vmax.z = std::max(vmax.z, coords.z);

        float atomRad = radii[a];
        *maxAtom = std::max(*maxAtom, atomRad);
    }
    *minVal = vmin;
    *maxVal = vmax;
}


float4 *getArrayAtomPosRad(chain *C, unsigned int N) {
    float4 *result = new float4[N];
    atom *A = NULL;
    int id = 0;
    A = &C->residues[0].atoms[0];
    float3 coords;
    while (A != NULL) {
        coords = A->coor;

        float atomRad = radiusDic[A->element[0]];
        result[id].x = coords.x;
        result[id].y = coords.y;
        result[id].z = coords.z;
        result[id].w = atomRad;
        id++;
        A = A->next;
    }

    return result;
}


float4 *getArrayAtomPosRad(pdb *P, unsigned int N) {
    chain *C = NULL;
    atom *A = NULL;
    float4 *result = new float4[N];
    // float4 *result;
    int id = 0;

    for (int chainId = 0; chainId < P->size; chainId++) {
        C = &P->chains[chainId];

        A = &C->residues[0].atoms[0];
        float3 coords;
        while (A != NULL) {
            coords = A->coor;

            float atomRad = radiusDic[A->element[0]];
            result[id].x = coords.x;
            result[id].y = coords.y;
            result[id].z = coords.z;
            result[id].w = atomRad;
            id++;
            A = A->next;
        }
    }
    return result;
}


float4 *getArrayAtomPosRad(float3 *positions, float *radii, unsigned int N) {
    float4 *result = (float4 *)malloc(sizeof(float4) * N);
    int id = 0;

    for (int a = 0; a < N; a++) {
        float3 coords = positions[a];
        float atomRad = radii[a];
        result[id].x = coords.x;
        result[id].y = coords.y;
        result[id].z = coords.z;
        result[id].w = atomRad;
        id++;
    }
    return result;
}


float computeMaxDist(float3 minVal, float3 maxVal, float maxAtomRad) {
    return std::max(maxVal.x - minVal.x, std::max(maxVal.y - minVal.y, maxVal.z - minVal.z)) + (2 * maxAtomRad) + (4 * probeRadius);
}

void writeToObj(const string &fileName, const vector<int> &meshTriSizes, const vector<int> &meshVertSizes,
                const vector<float3*> &Allvertices, const vector<int3*> &AllTriangles) {

#if MEASURETIME
    std::clock_t start = std::clock();
#endif

    FILE *fptr;
    if ((fptr = fopen(fileName.c_str(), "w")) == NULL) {
        fprintf(stderr, "Failed to open output file\n");
        exit(-1);
    }
    for (int m = 0; m < meshTriSizes.size(); m++) {
        for (int i = 0; i < meshVertSizes[m]; i++) {
            float3 vert = Allvertices[m][i];
            fprintf(fptr, "v %.3f %.3f %.3f\n", vert.x, vert.y, vert.z);
        }
    }

    fprintf(fptr, "\n");
    unsigned int cumulMesh = 0;
    for (int m = 0; m < meshTriSizes.size(); m++) {
        int ntri = meshTriSizes[m];
        for (int i = 0; i < ntri; i++) {
            if(AllTriangles[m][i].x != AllTriangles[m][i].y && AllTriangles[m][i].x != AllTriangles[m][i].z && AllTriangles[m][i].y != AllTriangles[m][i].z){
                fprintf(fptr, "f %d %d %d\n", cumulMesh + AllTriangles[m][i].y + 1, cumulMesh + AllTriangles[m][i].x + 1, cumulMesh + AllTriangles[m][i].z + 1);
            }
        }
        cumulMesh += meshVertSizes[m];
    }

    fclose(fptr);

#if MEASURETIME
    std::cerr << "Time for writting " << (std::clock() - start) / (double)(CLOCKS_PER_SEC / 1000) << " ms" << std::endl;
#endif
}


void writeToPly(const string &fileName, std::vector<MeshData> meshes) {
  FILE *fptr;
  if ((fptr = fopen(fileName.c_str(), "w")) == NULL) {
    fprintf(stderr, "Failed to open output file\n");
    exit(-1);
  }
  fprintf(fptr, "ply\n");
  fprintf(fptr, "format ascii 1.0\n");
  fprintf(fptr, "comment author: Yang Zhang (y.zhang@bioc.uzh.ch)\n");
  fprintf(fptr, "element vertex %d\n", meshes[0].NVertices);
  fprintf(fptr, "property float x\n");
  fprintf(fptr, "property float y\n");
  fprintf(fptr, "property float z\n");
  fprintf(fptr, "element face %d\n", meshes[0].NTriangles);
//  fprintf(fptr, "property list uchar int vertex_index\n");
  fprintf(fptr, "property list uchar int vertex_indices\n");

  fprintf(fptr, "end_header\n");
  unsigned int cumulVert = 0;
  for (int m = 0; m < meshes.size(); m++) {
    MeshData mesh = meshes[m];
//    smoothMeshLaplacian(2, mesh);
    for (int i = 0; i < mesh.NVertices; i++) {
      float3 vert = mesh.vertices[i];
      fprintf(fptr, "%.3f %.3f %.3f\n", vert.x, vert.y, vert.z);
    }
  }
  fprintf(fptr, "\n");
  for (int m = 0; m < meshes.size(); m++) {
    MeshData mesh = meshes[m];

    for (int i = 0; i < mesh.NTriangles; i++) {
      fprintf(fptr, "3 %d %d %d\n", cumulVert + mesh.triangles[i].y, cumulVert + mesh.triangles[i].x, cumulVert + mesh.triangles[i].z);
    }
    cumulVert += mesh.NVertices;
  }
  fclose(fptr);
}


void writeToObj(const string &fileName, const MeshData &mesh) {
#if MEASURETIME
    std::clock_t start = std::clock();
#endif

    FILE *fptr;
    if ((fptr = fopen(fileName.c_str(), "w")) == NULL) {
        fprintf(stderr, "Failed to open output file\n");
        exit(-1);
    }

    for (int i = 0; i < mesh.NVertices; i++) {
        float3 vert = mesh.vertices[i];
        fprintf(fptr, "v %.3f %.3f %.3f\n", vert.x, vert.y, vert.z);

    }

    fprintf(fptr, "\n");
    for (int i = 0; i < mesh.NTriangles; i++) {
        fprintf(fptr, "f %d %d %d\n", mesh.triangles[i].y + 1, mesh.triangles[i].x + 1, mesh.triangles[i].z + 1);
    }
    fclose(fptr);
#if MEASURETIME
    std::cerr << "Time for writting " << (std::clock() - start) / (double)(CLOCKS_PER_SEC / 1000) << " ms" << std::endl;
#endif
}


void writeToObj(const string &fileName, std::vector<MeshData> meshes) {
#if MEASURETIME
    std::clock_t start = std::clock();
#endif

    FILE *fptr;
    if ((fptr = fopen(fileName.c_str(), "w")) == NULL) {
        fprintf(stderr, "Failed to open output file\n");
        exit(-1);
    }
    unsigned int cumulVert = 0;
    for (int m = 0; m < meshes.size(); m++) {
        MeshData mesh = meshes[m];

        // smoothMeshLaplacian(2, mesh);

        for (int i = 0; i < mesh.NVertices; i++) {
            float3 vert = mesh.vertices[i];
            fprintf(fptr, "v %.3f %.3f %.3f\n", vert.x, vert.y, vert.z );
        }
    }
    fprintf(fptr, "\n");
    for (int m = 0; m < meshes.size(); m++) {
        MeshData mesh = meshes[m];

        for (int i = 0; i < mesh.NTriangles; i++) {
            fprintf(fptr, "f %d %d %d\n", cumulVert + mesh.triangles[i].y + 1, cumulVert + mesh.triangles[i].x + 1, cumulVert + mesh.triangles[i].z + 1);
        }
        cumulVert += mesh.NVertices;
    }
    fclose(fptr);
#if MEASURETIME
    std::cerr << "Time for writting " << (std::clock() - start) / (double)(CLOCKS_PER_SEC / 1000) << " ms" << std::endl;
#endif
}


MeshData computeMarchingCubes(int3 sliceGridSESDim, int cutMC, int sliceNbCellSES, float *cudaGridValues, uint2* vertPerCell,
                              unsigned int *compactedVoxels, int3 gridSESDim, float4 originGridSESDx, int3 offset, float4 *cudaSortedAtomPosRad,
                              int2 *cellStartEnd, int3 gridNeighborDim, float4 originGridNeighborDx, int rangeSearchRefine) {
    
    unsigned long int memAlloc = 0;
    memsetCudaUInt2 <<< (sliceNbCellSES + NBTHREADS - 1) / NBTHREADS, NBTHREADS >>> (vertPerCell, make_uint2(0, 0), sliceNbCellSES);

    MeshData result;
    float iso = 0.0f;
    dim3 localWorkSize(cutMC, cutMC, cutMC);
    dim3 globalWorkSize((sliceGridSESDim.x + cutMC - 1) / cutMC, (sliceGridSESDim.y + cutMC - 1) / cutMC, (sliceGridSESDim.z + cutMC - 1) / cutMC);


    countVertexPerCell <<< globalWorkSize , localWorkSize >>>(iso, sliceGridSESDim, cudaGridValues, vertPerCell, rangeSearchRefine, offset);
    gpuErrchk( hipPeekAtLastError() );

    uint2 lastElement, lastScanElement;
    gpuErrchk(hipMemcpy((void *) &lastElement, (void *)(vertPerCell + sliceNbCellSES - 1), sizeof(uint2), hipMemcpyDeviceToHost));

    thrust::exclusive_scan(thrust::device_ptr<uint2>(vertPerCell),
                           thrust::device_ptr<uint2>(vertPerCell + sliceNbCellSES),
                           thrust::device_ptr<uint2>(vertPerCell),
                           make_uint2(0, 0), add_uint2());

    gpuErrchk(hipMemcpy((void *) &lastScanElement, (void *) (vertPerCell + sliceNbCellSES - 1), sizeof(uint2), hipMemcpyDeviceToHost));

    unsigned int totalVoxels = lastElement.y + lastScanElement.y;
    unsigned int totalVerts = lastElement.x + lastScanElement.x;

    float3 *cudaVertices;
    gpuErrchk(hipMalloc(&cudaVertices, sizeof(float3) * totalVerts));
    memAlloc += sizeof(float3) * totalVerts;

    globalWorkSize = dim3( (sliceGridSESDim.x + localWorkSize.x - 1) / localWorkSize.x, (sliceGridSESDim.y + localWorkSize.y - 1) / localWorkSize.y, (sliceGridSESDim.z + localWorkSize.z - 1) / localWorkSize.z );

    compactVoxels <<< globalWorkSize, localWorkSize>>>(compactedVoxels, vertPerCell, lastElement.y, sliceNbCellSES, sliceNbCellSES + 1, sliceGridSESDim, rangeSearchRefine, offset);
    gpuErrchk( hipPeekAtLastError() );

    unsigned int totalVoxsqr3 = (unsigned int )ceil((totalVoxels + NBTHREADS - 1) / NBTHREADS);
    globalWorkSize = dim3(totalVoxsqr3, 1, 1);
    if (totalVoxsqr3 == 0) {
        return result;
    }

    generateTriangleVerticesSMEM <<< globalWorkSize, NBTHREADS>>>(cudaVertices, compactedVoxels, vertPerCell, cudaGridValues, originGridSESDx,
            iso, totalVoxels, totalVerts - 3, sliceGridSESDim, offset);

    gpuErrchk( hipPeekAtLastError() );


    //Weld vertices
    float3 *vertOri;
    int *cudaTri;
    int *cudaAtomIdPerVert;

    int global = (unsigned int )ceil((totalVerts + NBTHREADS - 1) / NBTHREADS);
    groupVertices <<< global, NBTHREADS >>>(cudaVertices, totalVerts, EPSILON);
    gpuErrchk( hipPeekAtLastError() );

    gpuErrchk(hipMalloc(&vertOri, sizeof(float3) * totalVerts));
    gpuErrchk(hipMemcpy(vertOri, cudaVertices, sizeof(float3) * totalVerts, hipMemcpyDeviceToDevice));
    gpuErrchk(hipMalloc(&cudaTri, sizeof(int) * totalVerts));

    memAlloc += sizeof(float3) * totalVerts;
    memAlloc += sizeof(int) * totalVerts;
    

    thrust::device_ptr<float3> vertThrust(cudaVertices);
    thrust::sort(vertThrust, vertThrust + totalVerts, sort_float3());

    thrust::device_ptr<float3> last = thrust::unique(vertThrust, vertThrust + totalVerts, samefloat3());

    unsigned int newtotalVerts = last - vertThrust;

    thrust::device_ptr<float3> vertOriThrust(vertOri);
    thrust::device_ptr<int> triThrust(cudaTri);
    thrust::lower_bound(vertThrust, last, vertOriThrust, vertOriThrust + totalVerts, triThrust, lessf3<float3>());
    gpuErrchk( hipPeekAtLastError() );

    gpuErrchk(hipMalloc(&cudaAtomIdPerVert, sizeof(int) * newtotalVerts));
    memAlloc += sizeof(int) * newtotalVerts;

    global = (unsigned int )ceil((newtotalVerts + NBTHREADS - 1) / NBTHREADS);

    //Look for atoms around vertices => could be done a way smarter way during the MC step
    closestAtomPerVertex<<<global, NBTHREADS >>>(cudaAtomIdPerVert, cudaVertices, newtotalVerts, gridNeighborDim,
                                    originGridNeighborDx, originGridSESDx, cellStartEnd, cudaSortedAtomPosRad);

    gpuErrchk( hipPeekAtLastError() );

#if DEBUG_MODE
    cerr << "MC allocation = "<< memAlloc / 1000000.0f << " Mo" << endl;
#endif

    int Ntriangles = totalVerts / 3;

    result.vertices = (float3 *) malloc(sizeof(float3) * newtotalVerts);
    result.triangles = (int3 *) malloc(sizeof(int3) * Ntriangles);
    result.atomIdPerVert = (int *) malloc(sizeof(int) * newtotalVerts);
    result.NVertices = newtotalVerts;
    result.NTriangles = Ntriangles;

    int *tmpTri = (int *)malloc(sizeof(int) * totalVerts);

    gpuErrchk(hipMemcpy(result.vertices, cudaVertices, sizeof(float3)*newtotalVerts, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(result.atomIdPerVert, cudaAtomIdPerVert, sizeof(int) * newtotalVerts, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(tmpTri, cudaTri, sizeof(int)*totalVerts, hipMemcpyDeviceToHost));

    //Store the triangle in a 3d vector
    for (int i = 0; i < Ntriangles; i++) {
        result.triangles[i].x = tmpTri[i * 3 + 0];
        result.triangles[i].y = tmpTri[i * 3 + 1];
        result.triangles[i].z = tmpTri[i * 3 + 2];
    }
    free(tmpTri);

    gpuErrchk(hipFree(cudaVertices));
    gpuErrchk(hipFree(vertOri));
    gpuErrchk(hipFree(cudaTri));
    gpuErrchk(hipFree(cudaAtomIdPerVert));

    return result;
}


std::vector<MeshData> computeSlicedSES(float3 positions[], float radii[], unsigned int N, float resoSES, int doSmoothing = 1) {
#if MEASURETIME
    std::clock_t startSES = std::clock();
#endif

    //Record a mesh per slice
    std::vector<MeshData> resultMeshes;
    float3 minVal, maxVal;
    float maxAtomRad = 0.0;

    getMinMax(positions, radii, N, &minVal, &maxVal, &maxAtomRad);

    if (N <= 1) {
        cerr << "Failed to parse the PDB or empty PDB file" << endl;
        return resultMeshes;
    }

    float4 *atomPosRad = getArrayAtomPosRad(positions, radii, N);
    float maxDist = computeMaxDist(minVal, maxVal, maxAtomRad);

    gridResolutionNeighbor = probeRadius + maxAtomRad;

    //Grid is a cube
    float3 originGridNeighbor = {
        minVal.x - maxAtomRad - 2 * probeRadius,
        minVal.y - maxAtomRad - 2 * probeRadius,
        minVal.z - maxAtomRad - 2 * probeRadius
    };

    int gridNeighborSize = (int)ceil(maxDist / gridResolutionNeighbor);

    int3 gridNeighborDim = {gridNeighborSize, gridNeighborSize, gridNeighborSize};

    int gridSESSize = (int)ceil(maxDist / resoSES);

    int3 gridSESDim = {gridSESSize, gridSESSize, gridSESSize};

    float4 originGridNeighborDx = {
        originGridNeighbor.x,
        originGridNeighbor.y,
        originGridNeighbor.z,
        gridResolutionNeighbor
    };

    float4 originGridSESDx = {
        originGridNeighborDx.x,
        originGridNeighborDx.y,
        originGridNeighborDx.z,
        resoSES
    };

    unsigned int nbcellsNeighbor = gridNeighborDim.x * gridNeighborDim.y * gridNeighborDim.z;
    // unsigned int nbcellsSES = gridSESDim.x * gridSESDim.y * gridSESDim.z;

    float4 *cudaAtomPosRad;
    float4 *cudaSortedAtomPosRad;
    int2 *cudaHashIndex;
    int2 *cellStartEnd;
    float *cudaGridValues;
    int *cudaFillCheck;

    //Marching cubes data
    uint2* vertPerCell;
    unsigned int *compactedVoxels;

    gpuErrchk(hipMalloc((void **)&cudaAtomPosRad , sizeof(float4) * N));
    gpuErrchk(hipMalloc((void **)&cudaSortedAtomPosRad , sizeof(float4) * N));
    gpuErrchk(hipMalloc((void **)&cudaHashIndex, sizeof(int2) * N));
    gpuErrchk(hipMalloc((void**)&cellStartEnd, sizeof(int2) * nbcellsNeighbor));

    //-------------- Step 1 : Insert atoms in neighbor cells -----------------

    //Copy atom positions and radii to GPU
    gpuErrchk(hipMemcpy(cudaAtomPosRad, atomPosRad, sizeof(float4) * N, hipMemcpyHostToDevice));

    //Compute atom cell ids
    hashAtoms <<< N, NBTHREADS >>>(N, cudaAtomPosRad, gridNeighborDim, originGridNeighborDx, cudaHashIndex, N);

    gpuErrchk( hipPeekAtLastError() );

    //Sort atoms cell id
    compare_int2 cmp;
    thrust::device_ptr<int2> D_beg = thrust::device_pointer_cast(cudaHashIndex);
    thrust::sort(D_beg, D_beg + N, cmp);
    gpuErrchk( hipPeekAtLastError() );

    memsetCudaInt2 <<< (nbcellsNeighbor + NBTHREADS - 1) / NBTHREADS, NBTHREADS >>> (cellStartEnd, make_int2(EMPTYCELL, EMPTYCELL), nbcellsNeighbor);

    //Reorder atoms positions and radii and fill cellStartEnd
    sortCell <<< N , NBTHREADS>>>(N, cudaAtomPosRad, cudaHashIndex, cudaSortedAtomPosRad, cellStartEnd);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipFree(cudaAtomPosRad) );


    //-------------- Step 2 : Compute points of the grid outside or inside the surface -----------------
    //Use slices of the grid to avoid allocating large amount of data
    int rangeSearchRefine = (int)ceil(PROBERADIUS / resoSES);
    int sliceSmallSize = min(SLICE , gridSESSize);
    int sliceSize = min(SLICE + 2 * rangeSearchRefine, gridSESSize);
    // int sliceSmallNbCellSES = sliceSmallSize * sliceSmallSize * sliceSmallSize;
    int sliceNbCellSES = sliceSize * sliceSize * sliceSize;
    // int3 sliceGridSESDim = make_int3(sliceSmallSize, sliceSmallSize, sliceSmallSize);
    int3 fullSliceGridSESDim = make_int3(sliceSize, sliceSize, sliceSize);

    gpuErrchk(hipMalloc((void **)&cudaGridValues, sizeof(float) * sliceNbCellSES));
    gpuErrchk(hipMalloc((void **)&cudaFillCheck, sizeof(int) * sliceNbCellSES));

    gpuErrchk( hipMalloc(&vertPerCell, sizeof(uint2) * sliceNbCellSES) );
    gpuErrchk( hipMalloc(&compactedVoxels, sizeof(unsigned int) * sliceNbCellSES) );

    gpuErrchk( hipPeekAtLastError() );

#if DEBUG_MODE
    cerr << "#atoms : "<<N<<endl;
    cerr << "Allocating " << (( (sizeof(int) + sizeof(float)) * sliceNbCellSES + 3 * sizeof(int) * sliceNbCellSES) +  2 * sizeof(float4) * N +
                            sizeof(int2) * N + sizeof(int2) * nbcellsNeighbor )/ 1000000.0f << " Mo" << endl;
    cerr << "Full size grid = " << gridSESSize << " x " << gridSESSize << " x " << gridSESSize << endl;
#endif

    int3 offset = {0, 0, 0};
    int cut = 8;

    for (int i = 0; i < gridSESSize; i += sliceSmallSize) {
        offset.x = i;
        for (int j = 0; j < gridSESSize; j += sliceSmallSize) {
            offset.y = j;
            for (int k = 0; k < gridSESSize; k += sliceSmallSize) {
                offset.z = k;
                // cerr << "-----------------------------\nStarting : " << offset.x << " / " << offset.y << " / " << offset.z << endl;

                memsetCudaFloat <<< (sliceNbCellSES + NBTHREADS - 1) / NBTHREADS, NBTHREADS >>> (cudaGridValues, probeRadius, sliceNbCellSES);
                memsetCudaInt <<< (sliceNbCellSES + NBTHREADS - 1) / NBTHREADS, NBTHREADS >>> (cudaFillCheck, EMPTYCELL, sliceNbCellSES);

                dim3 localWorkSize(cut, cut, cut);
                // dim3 globalWorkSize((sliceSmallSize + cut - 1) / cut, (sliceSmallSize + cut - 1) / cut, (sliceSmallSize + cut - 1) / cut);
                dim3 globalWorkSize((sliceSize + cut - 1) / cut, (sliceSize + cut - 1) / cut, (sliceSize + cut - 1) / cut);

                int3 reducedOffset = make_int3(max(0, offset.x - rangeSearchRefine),
                                               max(0, offset.y - rangeSearchRefine),
                                               max(0, offset.z - rangeSearchRefine));

                probeIntersection <<< globalWorkSize, localWorkSize >>>(cudaFillCheck, cudaHashIndex, gridNeighborDim, originGridNeighborDx,
                        gridSESDim, fullSliceGridSESDim, originGridSESDx, cellStartEnd,
                        cudaSortedAtomPosRad, cudaGridValues, /*offset*/ reducedOffset, N, sliceNbCellSES);


                gpuErrchk( hipPeekAtLastError() );
                gpuErrchk( hipDeviceSynchronize() );

                //Count cells at the border, cells that will be used in the refinement step
                thrust::device_ptr<int> fillThrust(cudaFillCheck);
                thrust::sort(fillThrust, fillThrust + sliceNbCellSES);

                unsigned int notEmptyCells = thrust::count_if(thrust::device, fillThrust, fillThrust + sliceNbCellSES, is_notempty());


                if (notEmptyCells == 0) {
                    // cerr << "Empty cells !!!" << endl;
                    continue;
                }

                localWorkSize = dim3(NBTHREADS, 1.0f,  1.0f);

                //Too long execution of this kernel triggers the watchdog timer => cut it
                int tranche = min(notEmptyCells, 65536 / 8 * NBTHREADS);

                const int nbStream = 4;
                hipStream_t streams[nbStream];
                for (int i = 0; i < nbStream; i++)
                    hipStreamCreate(&(streams[i]));
                int idStream = 0;

                for (unsigned int o = 0; o < notEmptyCells; o += tranche) {

                    globalWorkSize = dim3((tranche + NBTHREADS - 1) / NBTHREADS, 1.0f, 1.0f);
                    // cerr <<o<< " Launch (" << globalWorkSize.x << ", "<<globalWorkSize.y<<", "<<globalWorkSize.z<<") x ("<<localWorkSize.x<<", "<<localWorkSize.y<<", 1.0)" << endl;

                    distanceFieldRefine <<< globalWorkSize, localWorkSize, 0, streams[idStream]>>> (cudaFillCheck, cudaHashIndex, gridNeighborDim, originGridNeighborDx,
                            gridSESDim, fullSliceGridSESDim, originGridSESDx, cellStartEnd,
                            cudaSortedAtomPosRad, cudaGridValues, N, notEmptyCells, reducedOffset, o);

                    idStream++;
                    if (idStream == nbStream)
                        idStream = 0;
                }

                gpuErrchk( hipPeekAtLastError() );
                gpuErrchk( hipDeviceSynchronize() );
                for (int i = 0; i < nbStream; i++)
                    hipStreamDestroy(streams[i]);
                //Reset grid values that are outside of the slice
                //Marching cubes
                MeshData mesh = computeMarchingCubes(fullSliceGridSESDim, cut, sliceNbCellSES, cudaGridValues,
                                                     vertPerCell, compactedVoxels, gridSESDim, originGridSESDx, reducedOffset,
                                                     cudaSortedAtomPosRad, cellStartEnd, gridNeighborDim, originGridNeighborDx, rangeSearchRefine);

                smoothMeshLaplacian(doSmoothing, mesh);
                resultMeshes.push_back(mesh);
            }
        }
    }

    hipFree(cudaSortedAtomPosRad);
    hipFree(cudaHashIndex);
    hipFree(cellStartEnd);
    hipFree(cudaGridValues);
    hipFree(cudaFillCheck);
    hipFree(vertPerCell);
    hipFree(compactedVoxels);

    free(atomPosRad);

#if MEASURETIME
    std::cout << "Time Measure: " << MEASURETIME << " ";
    std::cerr << "Time for computing SES " << (std::clock() - startSES) / (double)(CLOCKS_PER_SEC / 1000) << " ms" << std::endl;
#endif
    return resultMeshes;
}


extern "C"{
    int NTriangles;
    int NVertices;
    int *globalTriangles;
    float3 *globalVertices;
    int *globalIdAtomPerVert;
}


API void API_computeSES(float resoSES, float3 *atomPos, float *atomRad, unsigned int N, float3 *out_vertices,
    unsigned int *NVert, int *out_triangles, unsigned int *NTri, int doSmoothing) {
    *NVert = 0;
    *NTri = 0;
    std::vector<MeshData> resultMeshes = computeSlicedSES(atomPos, atomRad, N, resoSES, doSmoothing);

    unsigned int totalVerts = 0;
    unsigned int totalTris = 0;

    for (int i = 0; i < resultMeshes.size(); i++) {
        totalVerts += resultMeshes[i].NVertices;
        totalTris += resultMeshes[i].NTriangles*3;
    }
    globalVertices = (float3 *)malloc(sizeof(float3) * totalVerts);
    globalTriangles = (int *)malloc(sizeof(int) * totalTris);
    globalIdAtomPerVert = (int *)malloc(sizeof(int) * totalVerts);

    unsigned int cumulVert = 0;
    unsigned int curIdV = 0;
    unsigned int curIdT = 0;

    for (int i = 0;i < resultMeshes.size(); i++) {
        for (int v = 0; v < resultMeshes[i].NVertices; v++) {
            globalVertices[curIdV] = resultMeshes[i].vertices[v];
            globalIdAtomPerVert[curIdV] = resultMeshes[i].atomIdPerVert[v];
            curIdV++;
        }
        for(int t = 0; t < resultMeshes[i].NTriangles; t++){
            globalTriangles[curIdT++] = resultMeshes[i].triangles[t].x + cumulVert;
            globalTriangles[curIdT++] = resultMeshes[i].triangles[t].y + cumulVert;
            globalTriangles[curIdT++] = resultMeshes[i].triangles[t].z + cumulVert;
        }
        cumulVert += resultMeshes[i].NVertices;
    }

    *NVert = totalVerts;
    *NTri = totalTris;
    NTriangles = totalTris;
    NVertices = totalVerts;
}


extern "C"{
    API int *API_getTriangles(bool invertTriangles = false){
        if(invertTriangles){
            for(unsigned int t = 0; t < NTriangles / 3; t++){
                int save = globalTriangles[t * 3];
                globalTriangles[t * 3] = globalTriangles[t * 3 + 1];
                globalTriangles[t * 3 + 1] = save;
            }
        }
        return globalTriangles;
    }
    API float3 *API_getVertices(){
        return globalVertices;
    }
    API int *API_getAtomIdPerVert(){
        return globalIdAtomPerVert;
    }

    API void API_freeMesh() {
        free(globalVertices);
        free(globalTriangles);
        free(globalIdAtomPerVert);
    }
}


int main(int argc, const char * argv[]) {
    args::ArgumentParser parser("QuickSES, SES mesh generation using GPU", "");
    args::Group groupMandatory(parser, "", args::Group::Validators::All);
    args::Group groupOptional(parser,  "", args::Group::Validators::DontCare);
    args::ValueFlag<string> inFile(groupMandatory, "input.pdb", "Input PDB file", {'i'});
    args::ValueFlag<string> outFile(groupMandatory, "output.obj", "Output OBJ mesh file", {'o'});
    args::ValueFlag<int> smoothTimes(groupOptional, "smooth factor", "(1) Times to run Laplacian smoothing step.", {'l'});
    args::ValueFlag<float> voxelSize(groupOptional, "voxel size", "(0.5) Voxel size in Angstrom. Defines the quality of the mesh.", {'v'});
    args::ValueFlag<int> slice(groupOptional, "slice size", "(300) Size of the sub-grid. Defines the quantity of GPU memory needed.", {'s'});
    args::HelpFlag help(groupOptional, "help", "   Display this help menu", {'h', "help"});

    try {
        parser.ParseCLI(argc, argv);
    }
    catch (args::Help) {
        std::cerr << parser;
        return 0;
    }
    catch (args::ParseError e) {
        std::cerr << e.what() << std::endl;
        std::cerr << parser;
        return -1;
    }
    catch (args::ValidationError e) {
        // std::cerr << e.what() << std::endl;
        std::cerr << "Usage: " << parser;
        return -1;
    }

    if (inFile) { inputFilePath = args::get(inFile); }
    if (outFile) { outputFilePath = args::get(outFile); }
    if (smoothTimes) { laplacianSmoothSteps = args::get(smoothTimes); }
    if (voxelSize) { gridResolutionSES = args::get(voxelSize); }
    if (slice) {SLICE = args::get(slice); }

    std::clock_t startparse = std::clock();

    initRadiusDic();

    pdb *P;
    P = initPDB();

    parsePDB((char *)inputFilePath.c_str(), P, (char *)"");

    cerr << "Grid resolution = " << gridResolutionSES << endl;
    std::cerr << "Time for parse " << (std::clock() - startparse) / (double)(CLOCKS_PER_SEC / 1000) << " ms" << std::endl;

    unsigned int N = 0;
    std::vector<float3> atomPos;
    std::vector<float> atomRadii;

    atom *A = NULL;
    chain *C = NULL;

    for (int chainId = 0; chainId < P->size; chainId++) {
        C = &P->chains[chainId];
        A = &C->residues[0].atoms[0];
        while (A != NULL) {
            float3 coords = A->coor;
            atomPos.push_back(coords);
            float atomRad;
            if (radiusDic.count(A->element[0]))
                atomRad = radiusDic[A->element[0]];
            else
                atomRad = radiusDic['X'];
            atomRadii.push_back(atomRad);

            N++;
            A = A->next;
        }
    }

    std::vector<MeshData> resultMeshes = computeSlicedSES(&atomPos[0], &atomRadii[0], N, gridResolutionSES, laplacianSmoothSteps);
    // std::vector<MeshData> resultMeshes = computeSlicedSESCPU(P);

    //Write to OBJ
    writeToObj(outputFilePath, resultMeshes);
    // writeToPly(outputFilePath, resultMeshes);
    freePDB(P);
    return 0;
}

std::vector<MeshData> get_mesh_by_xyzr(float *ptr, int N, int M, float grid_spacing, int smooth_steps, int slice_size){
  std::vector<float3> atomPos;
  std::vector<float> atomRadii;
  for (int i = 0; i < N; i++){
    atomPos.push_back(make_float3(ptr[i*M], ptr[i*M+1], ptr[i*M+2]));
    atomRadii.push_back(ptr[i*M+3]);
  }
  SLICE = slice_size;
  std::vector<MeshData> resultMeshes = computeSlicedSES(&atomPos[0], &atomRadii[0], N, grid_spacing, smooth_steps);
  return resultMeshes;
}
